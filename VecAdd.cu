#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Threads to be run in parallel
#define CHECK_CUDA_ERROR(call)                                                                         \
    {                                                                                                  \
        hipError_t err = call;                                                                        \
        if (err != hipSuccess)                                                                        \
        {                                                                                              \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                                        \
        }                                                                                              \
    }

// Kernel definition
__global__ void VecAdd(float *A, float *B, float *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        C[i] = A[i] + B[i];
}

int main()
{
    // Vector size (number of elements)
    int N = 50;
    size_t size = N * sizeof(float);

    // Allocate host memory, can use vectors but cuda built on C so pointers to  be consistent.
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Check if memory allocation was successful
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Host memory allocation failed\n");
        exit(EXIT_FAILURE);
    }

    // Initialize host arrays
    for (int i = 0; i < N; i++)
    {
        h_A[i] = i;
        h_B[i] = i * 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_A, size));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_B, size));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_C, size));

    // Copy data from host to device
    CHECK_CUDA_ERROR(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    // <<<Blocks, Threads>>>
    int numThreads = 16;
    VecAdd<<<1, numThreads>>>(d_A, d_B, d_C, N);

    // Must copy result back from device (GPU) to host (CPU) before accessing
    CHECK_CUDA_ERROR(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

    for (int i = 0; i < numThreads; i++)
    {
        printf("h_C[%d] = %f\n", i, h_C[i]);
    }

    // Free the memory on the cuda device
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free the memory on the host device
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}