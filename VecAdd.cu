#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


// Kernel definition
__global__ void VecAdd(float *A, float *B, float *C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main()
{
    // Vector size
    const int N = 100;
    size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; i++)
    {
        h_A[i] = i;
        h_B[i] = i * 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy host memory to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Kernel invocation with N threads
    VecAdd<<<1, N>>>(d_A, d_B, d_C);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify results
    for (int i = 0; i < N; i++)
    {
        printf("h_C[%d] = %f\n", i, h_C[i]);
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}